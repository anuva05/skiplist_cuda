#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "skip_parallel.h"
#define MAX_LEVEL 32

#define MATCH_BARRIER 0x80000000

struct Node {
  Node **next;
  E val;
  int level;
};

struct Skiplist {
  Node *head;
};

/* utility functions: do not use outside of this source file */
__device__ static Node *node_search(Skiplist *sl, E elem, int desired_level);
__device__ static Node *node_create(E val, int level);
__device__ static void node_destroy(Node *node);
__global__ static void skiplist_size_internal(Skiplist *sl, int *size_out);
__global__ static void create_head(Skiplist *sl);
__global__ static void skiplist_destroy_traverse(Skiplist *sl, Node **to_free);
__global__ static void skiplist_destroy_free(Node **to_free, int size);
__device__ static int rand(unsigned int random);
__global__ static void skiplist_gather_internal(E *dest, Skiplist *sl);

/* Initializes the skiplist to contain nothing but an empty head node */
__global__ static void create_head(Skiplist *sl)
{
  sl->head = node_create(MIN_VAL, MAX_LEVEL);
  memset(sl->head->next, 0, MAX_LEVEL * sizeof(Node *));
}

/* Performs a traversal of the list and completes preliminary legwork so that
 * memory can be freed after it is used in the list.
 */
__global__ static void skiplist_destroy_traverse(Skiplist *sl, Node **to_free)
{
  Node *cur;

  int i = 0;

  cur = sl->head;

  // traverse through the nodes and delete each of them from memory
  while (cur != NULL) {
    to_free[i] = cur;

    cur = cur->next[0];
    i++;
  }
}

/* Uses the array of elements created above to actually free up Node memory. */
__global__ static void skiplist_destroy_free(Node **to_free, int size)
{
  // Parallelizing the process of freeing Nodes from memory.
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  // Frees Nodes like in skip_serial but uses threads to do this concurrently
  while (i < size) {
    free(to_free[i]->next);
    free(to_free[i]);

    i += blockDim.x * gridDim.x;
  }
}

/* Launches a kernel to create a header for the new skiplist. */
Skiplist *skiplist_create(void)
{
  Skiplist *sl;
  CHECK(hipMalloc(&sl, sizeof(Skiplist)));

  create_head<<<1, 1>>>(sl);
  CHECK(hipDeviceSynchronize());

  return sl;
}

/* Combines destory_traverse and destroy_free functions to actually free
 * memory once the skiplist need not be used again.
 */
void skiplist_destroy(Skiplist *sl)
{
  Node **to_free;
  int size = skiplist_size(sl);

  CHECK(hipMalloc((void **)&to_free, size * sizeof(Node *)));

  skiplist_destroy_traverse<<<1, 1>>>(sl, to_free);
  skiplist_destroy_free<<<100, 512>>>(to_free, size);
  CHECK(hipDeviceSynchronize());
  CHECK(hipFree(sl));
  CHECK(hipFree(to_free));
}

/* Inserts a new node into the list in parallel. */
__device__ void skiplist_insert(Skiplist *sl, E elem)
{
  Node *new_node;
  Node *dest;
  Node *link_first_read, *link_second_read;
  int i;
  int level;

  // Randomly generate i.
  // The level of the node is given by the number of successive 1-bits at
  // the tail of i, plus 1.
  level = 1;
  while (rand(clock()) == 1 && level < MAX_LEVEL)
    level++;

  new_node = node_create(elem, level);

  // insert the new node into the skiplist
  //int tries;
  for (i = 0; i < level; i++) {
    do {
      // We keep track of the original form of the list for checking purposes
      dest = node_search(sl, elem, i); // want to insert right after this node
      link_first_read = dest->next[i];

      // Checks for obvious errors before inserting
      if (link_first_read != NULL && link_first_read->val < elem) {
        continue;
      }

      // Allow each node/thread to set the forward facing pointer.
      new_node->next[i] = link_first_read;
      // check if dest->next[i] contains the same value as a while ago
      // if so, make it point to the new node. 
      // otherwise, declare failure and try again.
      // this would mean that someone else has already inserted.
      link_second_read
      = (Node *)atomicCAS((unsigned long long int *)&(dest->next[i]),
        *(unsigned long long int *)&link_first_read,
        *(unsigned long long int *)&new_node);
    } while (link_first_read != link_second_read);
  }
}
 
/* Simply traverses the bottom level of the skiplist and returns size */
int skiplist_size(Skiplist *sl)
{
  int *size, *size_dev;
  int size_result;
  CHECK(hipHostAlloc(&size, sizeof(int), hipHostMallocMapped));
  CHECK(hipHostGetDevicePointer(&size_dev, size, 0));

  skiplist_size_internal<<<1, 1>>>(sl, size_dev);
  CHECK(hipDeviceSynchronize());

  size_result = *size;
  CHECK(hipHostFree(size));

  return size_result;
}
/* Retrieves the size on the device and writes back to the kernel above. */
__global__ static void skiplist_size_internal(Skiplist *sl, int *size_out)
{
  Node *cur = skiplist_head(sl);
  int size = 0;

  if (cur->next[0] == NULL) {
    *size_out = 0;
    return;
  }

  cur = cur->next[0]; // skip the (empty) head node

  while (cur != NULL) {
    size++;
    cur = cur->next[0];
  }

  *size_out = size; // write across PCI channel
}

/* Returns an array of type E containing only sorted values from the list */
E *skiplist_gather(Skiplist *sl, int *dim)
{
  int size = skiplist_size(sl);
  E *dest_dev;
  E *dest = (E *)malloc(size * sizeof(E));

  CHECK(hipMalloc(&dest_dev, size * sizeof(E)));

  skiplist_gather_internal<<<1, 1>>>(dest_dev, sl);
  CHECK(hipDeviceSynchronize());
  CHECK(hipMemcpy(dest, dest_dev, size * sizeof(E), hipMemcpyDeviceToHost));
  CHECK(hipFree(dest_dev));

  *dim = size;
  return dest;
}

/* Retrieves the elements of the list on the device much like size_internal. */
__global__ static void skiplist_gather_internal(E *dest, Skiplist *sl)
{
  Node *cur = skiplist_head(sl);
  int i;

  if (cur->next[0] == NULL)
    return;

  cur = cur->next[0];

  i = 0;
  while (cur != NULL) {
    dest[i] = cur->val;
    cur = cur->next[0];
    i++;
  }
}

/* Returns the pointer to the head of the skip list. */
__device__ Node *skiplist_head(Skiplist *sl)
{
  return sl->head;
}

/* Fetches the reference to the next Node in the skiplist. */
__device__ Node *node_next(Node *node)
{
  if (node == NULL)
    return NULL;
  else
    return node->next[0];
}

/* Constructor function for a single node belonging to the skiplist. */
__device__ static Node *node_create(E val, int level)
{
  Node *node = (Node *)malloc(sizeof(Node));

  node->val = val;
  node->level = level;
  node->next = (Node **)malloc(level * sizeof(Node *));

  return node;
}

/* Simplifies the process of freeing node data. */
__device__ static void node_destroy(Node *node)
{
  free(node->next);
  free(node);
}

/* Traverses the skiplist and searches for a particular element. */
__device__ static Node *node_search(Skiplist *sl, E elem, int desired_level)
{
  Node *cur = skiplist_head(sl);
  Node *next_node;
  int level;

  for (level = MAX_LEVEL - 1; level >= desired_level; level--) {
    next_node = cur->next[level];
    while (next_node != NULL && next_node->val < elem) {
      cur = next_node;
      next_node = cur->next[level];
    }
  }

  return cur;
}
/* Necessary function for generating a random number on the GPU. This is  
 * utilized when creating a new node and assigning an appropriate level value.
 */
__device__ static int rand(unsigned int random)
{
  //See Figure 2 of 'GPU Random Numbers via the Tiny Encryption Algorithm', Zafar (2010).
  unsigned int sum, v0, v1;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  //Start hashing.
  sum = 0;
  v0 = tid;
  v1 = random;
  
  sum += 0x9e3779b9;
  v0 += ((v1 << 4) + 0xa341316c)^(v1 + sum)^((v1 >> 5) + 0xc8013ea4);
  v1 += ((v0 << 4) + 0xad90777d)^(v0 + sum)^((v0 >> 5) + 0x7e95761e);
  
  sum += 0x9e3779b9;
  v0 += ((v1 << 4) + 0xa341316c)^(v1 + sum)^((v1 >> 5) + 0xc8013ea4);
  v1 += ((v0 << 4) + 0xad90777d)^(v0 + sum)^((v0 >> 5) + 0x7e95761e);
  
  sum += 0x9e3779b9;
  v0 += ((v1 << 4) + 0xa341316c)^(v1 + sum)^((v1 >> 5) + 0xc8013ea4);
  v1 += ((v0 << 4) + 0xad90777d)^(v0 + sum)^((v0 >> 5) + 0x7e95761e);
  
  sum += 0x9e3779b9;
  v0 += ((v1 << 4) + 0xa341316c)^(v1 + sum)^((v1 >> 5) + 0xc8013ea4);
  v1 += ((v0 << 4) + 0xad90777d)^(v0 + sum)^((v0 >> 5) + 0x7e95761e);
  
  return ((v0 + v1) < MATCH_BARRIER ? 0 : 1);
}
