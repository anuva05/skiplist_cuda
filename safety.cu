
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void _check(hipError_t cs, const char *file, long line)
{
  const char *errstr;

  if (cs != hipSuccess) {
    errstr = hipGetErrorString(cs);
    printf("CUDA error %s at %s:%ld.\n", errstr, file, line);
    exit(1);
  }
}
